#include "hip/hip_runtime.h"
#include "test_gpu.hpp"
#include "matrix_tools_gpu.cuh"

// #include "segment_gpu.cuh"
#include <cassert>
#include <vector>
#include <iostream>


[[gnu::noinline]]
void _abortError(const char* msg, const char* fname, int line)
{
  hipError_t err = hipGetLastError();
  std::cout << msg << " (" << fname << ", line: " << line << ")\n";
  std::cout << "Error " << hipGetErrorName(err) << ": " << hipGetErrorString(err) << "\n";
  std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

__global__ void mykernel(unsigned char* buffer, int width, int height, size_t pitch, 
                         unsigned char* outBuffer, size_t out_pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    outBuffer[out_pitch * y + x] = (x + buffer[pitch * y + x]) / 2;
}


void test_gpu(uint8_t* hostBuffer, int width, int height)
{
    hipError_t rc = hipSuccess;

    // alloc device memory
    unsigned char* devInputBuffer;
    size_t in_pitch;
    unsigned char* devOutputBuffer;
    size_t out_pitch;

    rc = hipMallocPitch(&devInputBuffer, &in_pitch, width * sizeof(uint8_t), height);
    if (rc)
        abortError("Fail buffer alloc");
    rc = hipMallocPitch(&devOutputBuffer, &out_pitch, width * sizeof(uint8_t), height);
    if (rc)
        abortError("Fail buffer alloc");
    rc = hipMemcpy2D(devInputBuffer, in_pitch, hostBuffer, width*sizeof(uint8_t), width * sizeof(uint8_t),
        height, hipMemcpyHostToDevice);
    if (rc)
        abortError("Cpy host to device fail");

    // run the kernel with blocks of size 64 * 64

    
    hipDeviceProp_t prop;
    hipGetDeviceProperties( &prop, 0);

    std::cout << "max nb_threads is " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "max threads dim is : " << prop.maxThreadsDim[0] << ", "
                                         << prop.maxThreadsDim[1] << ", "
                                         << prop.maxThreadsDim[2] << std::endl;
 
    int bsize = 32;
    int w = std::ceil((float)width / bsize);
    int h = std::ceil((float)height / bsize);

    std::cout << "running kernel of size " << w << " , " << h << std::endl;
    dim3 dimBlock(bsize, bsize);
    dim3 dimGrid(w, h);

    mykernel<<<dimGrid, dimBlock>>>(devInputBuffer, width, height,
                                    in_pitch, devOutputBuffer, out_pitch);
    if (hipPeekAtLastError())
        abortError("Computation Error");
    

    rc = hipMemcpy2D(hostBuffer, width * sizeof(uint8_t), devOutputBuffer, out_pitch, width * sizeof(char), height, hipMemcpyDeviceToHost);
      if (rc)
    abortError("Unable to copy buffer back to memory");

    // Free
    rc = hipFree(devInputBuffer);
    if (rc)
        abortError("Unable to free memory");
    rc = hipFree(devOutputBuffer);
    if (rc)
        abortError("Unable to free memory");
}


__device__ void predict(Filter* f)
  {
    using namespace kalman_gpu;

    float a[16] = {1, 1, 0, 0, 0, 1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1};
    kMatrix<float, 4, 4> A = kMatrix<float, 4, 4>(a);
    float a_t[16] = {1, 0, 0, 0, 1, 1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1};
    kMatrix<float, 4, 4> A_transpose(a_t);
    float c[12] = {1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1};
    kMatrix<float, 3, 4> C(c);
    float c_t[12] = {1, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1};
    kMatrix<float, 4, 3> C_transpose(c_t);
    float vn[9] = {2, 0, 0, 0,
                   1, 0, 0, 0, 12};
    kMatrix<float, 3, 3> Vn(vn);

    
    f->S_predicted = kMatrix<float, 4, 1>();
    matmul(A, f->S, f->S_predicted);
    add(f->S_predicted, f->W, f->S_predicted);

    f->X_predicted = kMatrix<float, 3, 1>();
    matmul(C, f->S_predicted, f->X_predicted);
    add(f->X_predicted, f->N, f->X_predicted);


    // f.S_predicted = A * f.S + f.W;
    // f.X_predicted = C * f.S_predicted + f.N;

    // uint32_t thik_d2 = f.X_predicted(1, 0) / 2;
    // f.n_min = f.X_predicted(0, 0) - thik_d2;
    // f.n_max = f.X_predicted(0, 0) + thik_d2;

    matmul(f->H, A_transpose, f->H);
    matmul(A, f->H, f->H);

    // f.H = A * f.H * A_transpose;

    f->W.buffer[0] = 0;
    f->W.buffer[1] = 0;

    f->obs_index = -1;
  }


__global__ void update_filters(float* obs_buffer, int* obs_count, int col, int max_height,
                               Filter* filter_buffer, int* integrations_buffer, int integration_padding)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    int nb_obs_in_col = obs_count[col + 1] - obs_count[col];

    if (x >= max_height)
        return;


    Filter* f = filter_buffer + x;
    int* integrations = integrations_buffer + (x * integration_padding);
    (void) integrations;
    (void) nb_obs_in_col;
    predict(f);

    // for (int i = 0; i < nb_obs_in_col; i++)
    // {
    // printf("col n%d, obs n%d, it has position of %f, thickness of %f and lum of %f\n",
    //         col,
    //         i, obs_buffer[obs_count[col] + i * 3],
    //         obs_buffer[obs_count[col] + i * 3 + 1],
    //         obs_buffer[obs_count[col] + i * 3 + 2]);
    // }
}

void traversal_gpu(float* obsHostBuffer, int* obsCount, int width, int max_height, int nb_obs)
{
    hipError_t rc = hipSuccess;

    // alloc device memory
    float* obs_buffer;
    int* obs_count_buffer;

    int integration_padding = nb_obs * width;
    std::vector<Filter> filter_host_buffer = std::vector<Filter>(nb_obs);
    std::vector<int> integrations_host_buffer = std::vector<int>(nb_obs * width, -1);
    int* integrations_device_buffer;
    Filter* filter_device_buffer;
    int nb_active_filters;

    rc = hipMalloc(&obs_buffer, nb_obs * sizeof(float) * 3);
    if (rc)
        abortError("Fail buffer alloc");
    rc = hipMalloc(&obs_count_buffer, width * sizeof(int));
    if (rc)
        abortError("Fail buffer alloc");
    rc = hipMalloc(&filter_device_buffer, nb_obs * sizeof(Filter));
    if (rc)
        abortError("Cuda Malloc fail");
    rc = hipMalloc(&integrations_device_buffer, nb_obs * width * sizeof(int));
    if (rc)
        abortError("Cuda Malloc fail");
    

    for (int i = 0; i < obsCount[0]; i++)
    {
        filter_host_buffer[i] = Filter(obs_buffer[i * 3],
                                       obs_buffer[i * 3 + 1],
                                       obs_buffer[i * 3 + 2]);
    }
    nb_active_filters = obsCount[0];

    rc = hipMemcpy(obs_buffer, obsHostBuffer, nb_obs * sizeof(float) * 3,
                hipMemcpyHostToDevice);
    if (rc)
        abortError("Cpy host to device fail");
    rc = hipMemcpy(obs_count_buffer, obsCount, width * sizeof(int),
            hipMemcpyHostToDevice);
    if (rc)
        abortError("Cpy host to device fail");
    rc = hipMemcpy(filter_device_buffer, filter_host_buffer.data(), nb_obs * sizeof(Filter),
        hipMemcpyHostToDevice);
    if (rc)
        abortError("Cpy host to device fail");
    rc = hipMemcpy(integrations_device_buffer, integrations_host_buffer.data(), nb_obs * width * sizeof(int),
        hipMemcpyHostToDevice);
    if (rc)
        abortError("Cpy host to device fail");

    // hipDeviceProp_t prop;
    // hipGetDeviceProperties( &prop, 0);

    // std::cout << "max nb_threads is " << prop.maxThreadsPerBlock << std::endl;
    // std::cout << "max threads dim is : " << prop.maxThreadsDim[0] << ", "
    //                                      << prop.maxThreadsDim[1] << ", "
    //                                      << prop.maxThreadsDim[2] << std::endl;
 
    int bsize = 512;
    int h = std::ceil((float)max_height / (bsize));

    std::cout << "running kernel of size " << h << std::endl;

    // for (int i = 1; i < width; i++)
    // {
    int i = 1;
    update_filters<<<h, bsize>>>(obs_buffer, obs_count_buffer, i,
                                        nb_active_filters, filter_device_buffer,
                                        integrations_device_buffer, integration_padding);
    // }
   
    if (hipPeekAtLastError())
        abortError("Computation Error");
    

    // rc = hipMemcpy2D(hostBuffer, width * sizeof(uint8_t), devOutputBuffer, out_pitch, width * sizeof(char), height, hipMemcpyDeviceToHost);
    //   if (rc)
    // abortError("Unable to copy buffer back to memory");

    // Free
    rc = hipFree(obs_buffer);
    if (rc)
        abortError("Unable to free memory");
    rc = hipFree(obs_count_buffer);
    if (rc)
        abortError("Unable to free memory");
}