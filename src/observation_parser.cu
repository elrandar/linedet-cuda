#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <err.h>
#include "observation_parser.hh"
#include "observation_parser.cuh"

namespace kalman {

/*
    {
        auto max = -1;
        auto start = -1;
        auto in_obs = false;

        if (max != -1)
            tmp_vec.emplace_back(static_cast<double>((start + height) / 2),
                                static_cast<double>(height - start),
                                static_cast<double>(max));
        for (int i = 0; i < height; ++i) {
            if (img[i * width + j] < threshold) {
                if (img[i * width + j] > max) {
                    max = img[i * width + j];
                }
                if (!in_obs) {
                    start = i;
                    in_obs = true;
                }
            }
            else {
                if (max != -1) {
                    tmp_vec.emplace_back(static_cast<double>((start + i) / 2),
                                        static_cast<double>(unsigned i - start),
                                        static_cast<double>(max));
                }
                max = -1;
                in_obs = false;
            }
        }
        vec.emplace_back(tmp_vec);
    }
    */

    __global__ void test_kernel(int* a) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        a[i] = 1;
    }

    __global__ void column_parser(int width, int height, u_int8_t *img, Eigen::Vector3d *vec, int threshold, unsigned int *sizes)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= width)
            return;

        auto vec_size = i;

        auto max = -1;
        auto start = -1;
        auto in_obs = false;

        if (max != -1) {
            vec[vec_size] = Eigen::Vector3d(static_cast<double>((start + height) / 2),
                                         static_cast<double>(height - start),
                                         static_cast<double>(max));
            vec_size += width;
        }

        for (int j = 0; j < height; ++j) {
            auto index = j * width + i;
            auto val_at = img[index];

            if (val_at < threshold) {
                if (val_at > max) {
                    max = val_at;
                }
                if (!in_obs) {
                    start = j;
                    in_obs = true;
                }
            }
            else {
                if (max != -1) {
                    vec[vec_size] = Eigen::Vector3d(static_cast<double>((start + j) / 2),
                                        static_cast<double>(j - start),
                                        static_cast<double>(max));
                    vec_size += width;
                }
                max = -1;
                in_obs = false;
            }
        }

        sizes[i] = (vec_size - i) / width;
    }

    std::pair<Eigen::Vector3d *, unsigned int*>
    obs_parser::parse_gpu(int width, int height, std::vector<u_int8_t> &img_host, int threshold)
    {
        auto size = width * height;
        // TODO is it better to gow with a vecotr of array of size height ??
        // TODO Try inplace with one array
        // TODO Try line by line
        // TODO Check pitch

        // Allocating memory for result array
        Eigen::Vector3d *vec = nullptr;
        auto err = hipMalloc(&vec, sizeof(Eigen::Vector3d) * size);
        if (err)
            errx(1, "Cuda vec malloc error code %d", err);

        // Allocating memory for image
        u_int8_t *img = nullptr;
        err = hipMalloc(&img, sizeof(u_int8_t) * size);
        if (err)
            errx(1, "Cuda img malloc error code %d", err);

        // Allocating res sizes for each column
        unsigned int *sizes = nullptr;
        err = hipMalloc(&sizes, sizeof(unsigned int) * width);
        if (err)
            errx(1, "Cuda sizes malloc error code %d", err);

        // Copying image into gpu buffer
        err = hipMemcpy(img, (u_int8_t*)(&img_host[0]), size * sizeof(u_int8_t), hipMemcpyHostToDevice);
        if (err)
            errx(1, "Cuda img-host memcpy error code %d", err);

        // Getting gpu specs
        int devId = 0;
        // There may be more devices!
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, devId);
        int xThreads = deviceProp.maxThreadsDim[0];
        dim3 DimBlock(xThreads, 1, 1);// 1D VecAddint
        auto xBlocks = (int) ceil(width / xThreads);
        dim3 DimGrid(xBlocks, 1, 1);

        // Calling kernel *width* times
        column_parser<<<1024, 1024>>>(width, height, img, vec, threshold, sizes);

        // Creating result array
        Eigen::Vector3d *res = new Eigen::Vector3d[size];
        // Copying res onto CPU
        err = hipMemcpy(res, vec, size * sizeof(Eigen::Vector3d), hipMemcpyDeviceToHost);
        if (err)
            errx(1, "Cuda vec-host memcpy error code %d", err);

        // Creating result sizes array
        unsigned int *host_sizes = new unsigned int[width];
        // Copying res sizes onto CPU
        err = hipMemcpy(host_sizes, sizes, width * sizeof(unsigned int), hipMemcpyDeviceToHost);
        if (err)
            errx(1, "Cuda sizes-host memcpy error code %d", err);

        std::cout << "Parse with GPU done.\n";

        return std::make_pair(res, host_sizes);
    }

    std::vector<std::vector<Eigen::Vector3d>> obs_parser::parse(int width, int height, std::vector<u_int8_t> img, int threshold)
    {
        std::vector<std::vector<Eigen::Vector3d>> vec;
        for(int j = 0; j < width; j++){
            std::vector<Eigen::Vector3d> tmp_vec;
            auto max = -1;
            //auto pos_max = -1;
            auto start = -1;
            auto in_obs = false;
            for(int i = 0; i < height; i++) {
                if (img[i * width + j] < threshold) {
                    if (img[i * width + j] > max) {
                        max = img[i * width + j];
                    }
                    if (!in_obs) {
                        start = i;
                        in_obs = true;
                    }
                }
                else {
                    if (max != -1) {
                        tmp_vec.push_back({static_cast<double>((start + i) / 2),
                                            static_cast<double>(i - start),
                                            static_cast<double>(max)});
                    }
                    max = -1;
                    //pos_max = -1;
                    in_obs = false;
                }
            }
            if (max != -1)
                tmp_vec.push_back({static_cast<double>((start + height) / 2),
                                    static_cast<double>(height - start),
                                    static_cast<double>(max)});
            vec.push_back(tmp_vec);
        }
        return vec;
    }

    void test_gpu(int *a) {
        int *a_device;

        auto err = hipMalloc(&a_device, 100 * sizeof(int));
        if (err)
            errx(1, "Cuda malloc error code %d", err);

        hipMemcpy(a_device, a, 100 * sizeof(int), hipMemcpyHostToDevice);

        test_kernel<<<100, 1>>>(a_device);

        hipMemcpy(a, a_device, 100 * sizeof(int), hipMemcpyDeviceToHost);

        hipFree(a_device);
    }
}
